#include "hip/hip_runtime.h"
#include <iostream>

#include "../headers/backgroundSub.hpp"
#include "../headers/pointsFinder.hpp"
#include "../headers/pointsRelation.hpp"


using namespace std;

// CUDA kernel. Each thread takes care of one element of c
__global__ void subVector(int *a, int *b, int *d, int *N){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid<*N) {
        d[tid] = a[tid] - b[tid];
    }
}

int main(int argc, char *argv[]){
    //vendo se o numero de argumento esta correto
    if(argc < 3){
        cout << "Erro nos parametros" << endl;
        cout << "Para utilizar:" << endl;
        cout << "./parallelTracking video" << endl;
    }

    Video video(argv[1]);
    Cuda cuda(video.imageSize());
    cuda.tansfer(video.ini_frame());
    //pegando o video enquanto tem quadros, a serem computados
    while(1){
        //se tem espaço em video
        if(cuda.hasMemory()){
            Video.hasframes();
            //movendo para a memoria de video

            //setando na lista de quadros a posic desta
        }
        //tem posic de memorias não computadas
        //assincrona

            //fazendo a subtração com o quadro anterior

            //pegando os pontos que se movimentotaram

            //fazendo a relação com os objetos anteriores

            //liberando imagens da memoria

        //puxando as relações da memoria de video
            //jogando em momoria

    //finalizado a contagem de tempo
    //imprimindo os resultados
    }
}
